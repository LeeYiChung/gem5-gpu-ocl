
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32
#define PADDING 2

__global__ void gaborFilterKernel(float* o_arr1d, float* i_arr1d, int rows, int cols, float* filter1d, int radius) {
    
    __shared__ float tile[BLOCK_SIZE+PADDING*2][BLOCK_SIZE+PADDING*2];
    __shared__ float filter_buf[2*PADDING+1][2*PADDING+1];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * blockDim.y + threadIdx.y;
    
    tile[ty+PADDING][tx+PADDING] = i_arr1d[yidx * cols + xidx];
    if(tx < 2*PADDING+1 && ty < 2*PADDING+1)
        filter_buf[ty][tx] = filter1d[ty * (2*PADDING+1) + tx];
    /*if(tx < PADDING)
        tile[ty+PADDING][tx] = i_arr1d[yidx * cols + (xidx - tx)];
    if(tx >= BLOCK_SIZE - PADDING)
        tile[ty+PADDING][tx+PADDING*2] = i_arr1d[yidx * cols + (xidx - tx + BLOCK_SIZE - 1)];
    if(ty < PADDING)
        tile[ty][tx+PADDING] = i_arr1d[(yidx - ty) * cols + xidx];
    if(ty >= BLOCK_SIZE - PADDING)
        tile[ty+PADDING*2][tx+PADDING] = i_arr1d[(yidx - ty + BLOCK_SIZE - 1) * cols + xidx];
    if(tx < PADDING && ty < PADDING)
        tile[ty][tx] = i_arr1d[(yidx - ty) * cols + xidx - tx];
    if(tx >= BLOCK_SIZE - PADDING && ty < PADDING)
        tile[ty][tx+PADDING*2] = i_arr1d[(yidx - ty) * cols + (xidx - tx + BLOCK_SIZE - 1)];
    if(tx < PADDING && ty >= BLOCK_SIZE - PADDING)
        tile[ty+PADDING*2][tx] = i_arr1d[(yidx - ty + BLOCK_SIZE - 1) * cols + xidx - tx];
    if(tx >= BLOCK_SIZE - PADDING && ty >= BLOCK_SIZE - PADDING)
        tile[ty+PADDING*2][tx+PADDING*2] = i_arr1d[(yidx - ty + BLOCK_SIZE - 1) * cols + xidx - tx + BLOCK_SIZE - 1];*/
    __syncthreads();

    float Io = 0.0f;
    float accumWeight = 0.0f;
    for(int dy = -radius; dy <= radius; dy++) {
        for(int dx = -radius; dx <= radius; dx++) {
            float weight = filter_buf[dy + radius][dx + radius];
            //float weight = filter1d[(dy + radius) * radius + (dx + radius)];
            float I = tile[ty + dy + PADDING][tx + dx + PADDING];
            Io += I * weight;
            accumWeight += weight;
        }
    }
    
    // normalize weight
    Io /= accumWeight;
    o_arr1d[yidx * cols + xidx] = Io;
    //o_arr1d[yidx * cols + xidx] = tile[ty+PADDING][tx+PADDING];
}
