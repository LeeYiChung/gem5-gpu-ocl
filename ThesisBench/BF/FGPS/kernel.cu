
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32
#define BLOCK_DEPTH 3
#define POW2(x) ((x) * (x))
__global__ void bilateralFilterKernel(float* o_arr1d, float* i_arr1d, int rows, int cols, int radius, float gauss_color_coeff, float gauss_space_coeff) {
    
    __shared__ float tile[BLOCK_DEPTH][BLOCK_SIZE][BLOCK_SIZE];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * blockDim.y + threadIdx.y;

    tile[0][ty][tx] = i_arr1d[yidx * cols + xidx];
    tile[1][ty][tx] = i_arr1d[rows * cols + yidx * cols + xidx];
    tile[2][ty][tx] = i_arr1d[2 * rows * cols + yidx * cols + xidx];

    __syncthreads();

    float Iir = tile[0][ty][tx];
    float Iig = tile[1][ty][tx];
    float Iib = tile[2][ty][tx];
    float Ior = 0.0f;
    float Iog = 0.0f;
    float Iob = 0.0f;
    float accumWeight = 0.0f;

    for(int dy = -radius; dy <= radius; dy++) {
        for(int dx = -radius; dx <= radius; dx++) {
            // test boundary of tile
            if(tx + dx >= 0 && tx + dx <= BLOCK_SIZE - 1 && ty + dy >= 0 && ty + dy <= BLOCK_SIZE - 1) {
                float Ir = tile[0][ty + dy][tx + dx];
                float Ig = tile[1][ty + dy][tx + dx];
                float Ib = tile[2][ty + dy][tx + dx];
                // intensity weight exponent
                float expi = gauss_color_coeff * (POW2(Ir - Iir) + POW2(Ig - Iig) + POW2(Ib - Iib));
                // spatial weight exponent
                float exps = gauss_space_coeff * (POW2(dx) + POW2(dy));
                
                float weight = expf(expi + exps);
                Ior += Ir * weight;
                Iog += Ig * weight;
                Iob += Ib * weight;
                accumWeight += weight;
            }
        }
    }

    // normalize weight
    Ior /= accumWeight;
    Iog /= accumWeight;
    Iob /= accumWeight;

    // write result to global memory
    o_arr1d[yidx * cols + xidx] = Ior;
    o_arr1d[rows * cols + yidx * cols + xidx] = Iog;
    o_arr1d[2 * rows * cols + yidx * cols + xidx] = Iob;

}
