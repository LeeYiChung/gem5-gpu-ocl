
#include <hip/hip_runtime.h>
#define BLOCK_WIDTH 32
//#define BLOCK_HEIGHT 32
#define BLOCK_HEIGHT 16
#define TILE_WIDTH 96
//#define TILE_HEIGHT 96
#define TILE_HEIGHT 48
#define SMALL_TILE_WIDTH 64
//#define SMALL_TILE_HEIGHT 64
#define SMALL_TILE_HEIGHT 32

__global__ void superResolutionKernel(float* o_arr1d, float* i_arr1d, int rows, int cols) {
    
    __shared__ float tile[TILE_HEIGHT][TILE_WIDTH];
    __shared__ float buf[SMALL_TILE_HEIGHT][TILE_WIDTH]; // buffer for down/up sampling
    __shared__ float small_tile[SMALL_TILE_HEIGHT][SMALL_TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    for(int offset_y = 0; offset_y < TILE_HEIGHT; offset_y += BLOCK_HEIGHT) {
        for(int offset_x = 0; offset_x < TILE_WIDTH; offset_x += BLOCK_WIDTH) {
            int xidx = blockIdx.x * TILE_WIDTH + offset_x + tx;
            int yidx = blockIdx.y * TILE_HEIGHT + offset_y + ty;
            tile[offset_y + ty][offset_x + tx] = i_arr1d[yidx * cols + xidx];
        }
    }
    __syncthreads();

    // col downsample: (TILE_HEIGHT, TILE_WIDTH) -> (SMALL_TILE_HEIGHT, TILE_WIDTH)
    for(int offset_x = 0; offset_x < TILE_WIDTH; offset_x += BLOCK_WIDTH) {
        buf[2*ty][offset_x + tx] = -0.022 * tile[3*ty][offset_x + tx] + 0.974 * tile[3*ty+1][offset_x + tx] + 0.227 * tile[3*ty+2][offset_x + tx];
        buf[2*ty+1][offset_x + tx] = 0.227 * tile[3*ty][offset_x + tx] + 0.974 * tile[3*ty+1][offset_x + tx] + -0.022 * tile[3*ty+2][offset_x + tx];
    }
    __syncthreads();

    // row downsample: (SMALL_TILE_HEIGHT, TILE_WIDTH) -> (SMALL_TILE_HEIGHT, SMALL_TILE_WIDTH)
    for(int offset_y = 0; offset_y < SMALL_TILE_HEIGHT; offset_y += BLOCK_HEIGHT) {
        small_tile[offset_y + ty][2*tx] = -0.022 * buf[offset_y + ty][3*tx] + 0.974 * buf[offset_y + ty][3*tx+1] + 0.227 * buf[offset_y + ty][3*tx+2];
        small_tile[offset_y + ty][2*tx+1] = 0.227 * buf[offset_y + ty][3*tx] + 0.974 * buf[offset_y + ty][3*tx+1] + -0.022 * buf[offset_y + ty][3*tx+2];
    }
    __syncthreads();

    // row upsample: (SMALL_TILE_HEIGHT, SMALL_TILE_WIDTH) -> (SMALL_TILE_HEIGHT, TILE_WIDTH)
    for(int offset_y = 0; offset_y < SMALL_TILE_HEIGHT; offset_y += BLOCK_HEIGHT) {
        // 3*tx
        if(tx == 0)
            buf[offset_y + ty][0] = 0.927 * small_tile[offset_y + ty][0] + -0.047 * small_tile[offset_y + ty][1];
        else
            buf[offset_y + ty][3*tx] = 0.119 * small_tile[offset_y + ty][2*tx-1] + 0.927 * small_tile[offset_y + ty][2*tx]
                + -0.047 * small_tile[offset_y + ty][2*tx+1];
        // 3*tx+2
        if(tx == BLOCK_WIDTH-1)
            buf[offset_y + ty][TILE_WIDTH-1] = -0.047 * small_tile[offset_y + ty][SMALL_TILE_WIDTH-2] + 0.927 * small_tile[offset_y + ty][SMALL_TILE_WIDTH-1];
        else
            buf[offset_y + ty][3*tx+2] = -0.047 * small_tile[offset_y + ty][2*tx] + 0.927 * small_tile[offset_y + ty][2*tx+1]
                + 0.119 * small_tile[offset_y + ty][2*tx+2];
        // 3*tx+1
        if(tx == 0)
            buf[offset_y + ty][1] = 0.6 * small_tile[offset_y + ty][0] + 0.6 * small_tile[offset_y + ty][1] + -0.1 * small_tile[offset_y + ty][2];
        else if(tx == BLOCK_WIDTH-1)
            buf[offset_y + ty][TILE_WIDTH-2] = -0.1 * small_tile[offset_y + ty][SMALL_TILE_WIDTH-3] + 0.6 * small_tile[offset_y + ty][SMALL_TILE_WIDTH-2] 
                + 0.6 * small_tile[offset_y + ty][SMALL_TILE_WIDTH-1];
        else
            buf[offset_y + ty][3*tx+1] = -0.1 * small_tile[offset_y + ty][2*tx-1] + 0.6 * small_tile[offset_y + ty][2*tx]
                + 0.6 * small_tile[offset_y + ty][2*tx+1] + -0.1 * small_tile[offset_y + ty][2*tx+2];
    }
    __syncthreads();

    // col upsample: (SMALL_TILE_HEIGHT, TILE_WIDTH) -> (TILE_HEIGHT, TILE_WIDTH)
    for(int offset_x = 0; offset_x < TILE_WIDTH; offset_x += BLOCK_WIDTH) {
        int xidx = blockIdx.x * TILE_WIDTH + offset_x + tx;
        // 3*ty
        if(ty == 0) {
            int yidx = blockIdx.y * TILE_HEIGHT;
            o_arr1d[yidx * cols + xidx] = 0.927 * buf[0][offset_x + tx] + -0.047 * buf[1][offset_x + tx];
        
        }
        else {
            int yidx = blockIdx.y * TILE_HEIGHT + 3*ty;
            o_arr1d[yidx * cols + xidx] = 0.119 * buf[2*tx-1][offset_x + tx] + 0.927 * buf[2*tx][offset_x + tx]
                + -0.047 * buf[2*tx+1][offset_x + tx];
        }
        // 3*ty+2
        if(ty == BLOCK_HEIGHT-1) {
            int yidx = blockIdx.y * TILE_HEIGHT + TILE_HEIGHT-1;
            o_arr1d[yidx * cols + xidx] = -0.047 * buf[SMALL_TILE_HEIGHT-2][offset_x + tx] + 0.927 * buf[SMALL_TILE_HEIGHT-1][offset_x + tx];
        }
        else {
            int yidx = blockIdx.y * TILE_HEIGHT + 3*ty+2;
            o_arr1d[yidx * cols + xidx] = -0.047 * buf[2*ty][offset_x + tx] + 0.927 * buf[2*ty+1][offset_x + tx]
                + 0.119 * small_tile[2*ty+2][offset_x + tx];
        }
        // 3*ty+1
        if(ty == 0) {
            int yidx = blockIdx.y * TILE_HEIGHT + 1;
            o_arr1d[yidx * cols + xidx] = 0.6 * buf[0][offset_x + tx] + 0.6 * buf[1][offset_x + tx] + -0.1 * buf[2][offset_x + tx];
        }
        else if(ty == BLOCK_HEIGHT-1) {
            int yidx = blockIdx.y * TILE_HEIGHT + TILE_HEIGHT-2;
            o_arr1d[yidx * cols + xidx] = -0.1 * buf[SMALL_TILE_HEIGHT-3][offset_x + tx] + 0.6 * buf[SMALL_TILE_HEIGHT-2][offset_x + tx] 
                + 0.6 * buf[SMALL_TILE_HEIGHT-1][offset_x + tx];
        }
        else {
            int yidx = blockIdx.y * TILE_HEIGHT + 3*ty+1;
            o_arr1d[yidx * cols + xidx] = -0.1 * buf[2*ty-1][offset_x + tx] + 0.6 * buf[2*ty][offset_x + tx]
                + 0.6 * buf[2*ty+1][offset_x + tx] + -0.1 * buf[2*ty+2][offset_x + tx];
        }
    }


}
