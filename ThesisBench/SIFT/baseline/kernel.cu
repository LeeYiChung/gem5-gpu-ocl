
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16
#define BLOCK_DEPTH 4

__global__ void KeypointLocalization(int* flags, float* DoG, int rows, int cols, float contrastThreshold, float curvateThreshold)
{
    __shared__ float tile[BLOCK_DEPTH][BLOCK_SIZE][BLOCK_SIZE];
    int tz = threadIdx.z;
    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int zidx = blockIdx.z * blockDim.z + threadIdx.z;
    int yidx = blockIdx.y * blockDim.y + threadIdx.y;
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = zidx * rows * cols + yidx * cols + xidx;
    
    tile[tz][ty][tx] = DoG[idx];
    __syncthreads();
    
    // exclude low contrast and boundary pixels
    if((tile[tz][ty][tx] > contrastThreshold) 
        && tx != 0 && tx != BLOCK_SIZE - 1
        && ty != 0 && ty != BLOCK_SIZE - 1
        && tz != 0 && tz != BLOCK_DEPTH - 1) {
        
        bool nExtrema = tile[tz][ty][tx] > tile[tz - 1][ty - 1][tx - 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz - 1][ty - 1][tx];
        nExtrema ^= tile[tz][ty][tx] > tile[tz - 1][ty - 1][tx + 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz - 1][ty][tx - 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz - 1][ty][tx];
        nExtrema ^= tile[tz][ty][tx] > tile[tz - 1][ty][tx + 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz - 1][ty + 1][tx - 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz - 1][ty + 1][tx];
        nExtrema ^= tile[tz][ty][tx] > tile[tz - 1][ty + 1][tx + 1];

        nExtrema ^= tile[tz][ty][tx] > tile[tz][ty - 1][tx - 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz][ty - 1][tx];
        nExtrema ^= tile[tz][ty][tx] > tile[tz][ty - 1][tx + 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz][ty][tx - 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz][ty][tx + 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz][ty + 1][tx - 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz][ty + 1][tx];
        nExtrema ^= tile[tz][ty][tx] > tile[tz][ty + 1][tx + 1];
        
        nExtrema ^= tile[tz][ty][tx] > tile[tz + 1][ty - 1][tx - 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz + 1][ty - 1][tx];
        nExtrema ^= tile[tz][ty][tx] > tile[tz + 1][ty - 1][tx + 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz + 1][ty][tx - 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz + 1][ty][tx];
        nExtrema ^= tile[tz][ty][tx] > tile[tz + 1][ty][tx + 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz + 1][ty + 1][tx - 1];
        nExtrema ^= tile[tz][ty][tx] > tile[tz + 1][ty + 1][tx];
        nExtrema ^= tile[tz][ty][tx] > tile[tz + 1][ty + 1][tx + 1];
    
        float Dxx = tile[tz][ty][tx - 1] + tile[tz][ty][tx + 1] - 2 * tile[tz][ty][tx];
        float Dyy = tile[tz][ty - 1][tx] + tile[tz][ty + 1][tx] - 2 * tile[tz][ty][tx];
        float Dxy = (tile[tz][ty + 1][tx + 1] - tile[tz][ty - 1][tx + 1] - tile[tz][ty + 1][tx - 1] + tile[tz][ty - 1][tx - 1]) / 4.0f;
        float Tr = Dxx + Dyy;
        float Det = Dxx * Dyy - Dxy * Dxy;
        float curvate = Tr * Tr / Det;
		
        flags[idx] = !nExtrema && Det >= 0 && curvate < curvateThreshold;
    }
}
