
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32
__global__ void opticalFlowKernel(int* o_gx, int* o_gy, int* arr1dI, int* arr1dJ, int rows, int cols, int wx, int wy, int K) {
    
    __shared__ int I_tile[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int J_tile[BLOCK_SIZE][BLOCK_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * blockDim.y + threadIdx.y;
   
    I_tile[ty][tx] = arr1dI[yidx * cols + xidx];
    J_tile[ty][tx] = arr1dJ[yidx * cols + xidx];

    __syncthreads();

    // Initialization of pyramidal guess
    float gx = 0, gy = 0;
    
    // Spatial gradient matrix
    float Gxx = 0, Gxy = 0, Gyy = 0;
    for(int dy = -wy; dy <= wy; dy++) {
        for(int dx = -wx; dx <= wx; dx++) {
            if(ty + dy - 1 >= 0 && ty + dy + 1 < BLOCK_SIZE && tx + dx - 1 >= 0 && tx + dx + 1 < BLOCK_SIZE) {
                int Ix = (I_tile[ty+dy][tx+dx+1] - I_tile[ty+dy][tx+dx-1]) / 2;
                int Iy = (I_tile[ty+dy+1][tx+dx] - I_tile[ty+dy-1][tx+dx]) / 2;
                Gxx += Ix * Ix;
                Gxy += Ix * Iy;
                Gyy += Iy * Iy;
            }
        }
    }

    float det = Gxx * Gyy - Gxy * Gxy;
    if(det > 0.00001f) {

        // Initialization of iterative L-K
        float vx = 0, vy = 0;
        for(int k = 0; k < K; k++) {
            int Jx = tx + gx + vx;
            int Jy = ty + gy + vy;
            if(Jx - 1 >= 0 && Jx + 1 < BLOCK_SIZE && Jy - 1 >= 0 && Jy + 1 < BLOCK_SIZE) {
                // Image mismatch vector
                float bkx = 0, bky = 0;
                for(int dy = -wy; dy <= wy; dy++) {
                    for(int dx = -wx; dx <= wx; dx++) {
                        if(ty + dy - 1 >= 0 && ty + dy + 1 < BLOCK_SIZE && tx + dx - 1 >= 0 && tx + dx + 1 < BLOCK_SIZE) {
                            int Ik = I_tile[ty+dy][tx+dx] - J_tile[Jy][Jx];
                            int Ix = (I_tile[ty+dy][tx+dx+1] - I_tile[ty+dy][tx+dx-1]) / 2;
                            int Iy = (I_tile[ty+dy+1][tx+dx] - I_tile[ty+dy-1][tx+dx]) / 2;
                            bkx += Ik * Ix;
                            bky += Ik * Iy;
                        }
                    }
                }
                // Optical Flow
                float eta_x = (-Gyy * bkx + Gxy * bky) / det;
                float eta_y = (Gxy * bkx - Gxx * bky) / det;
                
                // Guess for next iteration
                vx += eta_x;
                vy += eta_y;
            }
        }
        // Guess for next level
        gx += vx;
        gy += vy;
    }
    o_gx[yidx * cols + xidx] = gx;
    o_gy[yidx * cols + xidx] = gy;
}
